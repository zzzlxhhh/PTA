#include "hip/hip_runtime.h"

__device__ void reduceBlock2(qreal *arrayIn, qreal *reducedArray, int length) {
	int idx = threadIdx.x;

	// calculate lane index and warp index
	int laneIdx = threadIdx.x % warpSize;
	int warpIdx = threadIdx.x / warpSize;

	// blcok-wide warp reduce
	qreal localSum = warpReduce(arrayIn[idx]);

	// save warp sum to shared memory
	if (laneIdx == 0) arrayIn[warpIdx] = localSum;

	// block synchronization
	__syncthreads();

	//int size = ;
	// last warp reduce
	if (threadIdx.x < warpSize) localSum = (threadIdx.x < ((blockDim.x + 32 - 1) / 32)) ?
		arrayIn[laneIdx] : 0;

	if (warpIdx == 0)
		localSum = warpReduce(localSum);

	if (threadIdx.x == 0)
		reducedArray[blockIdx.x] = localSum;
}

__device__ void reduceBlock1(qreal *arrayIn, qreal *reducedArray, int length) {
	int i, l, r;
	int threadMax, maxDepth;
	threadMax = length / 2;
	maxDepth = log2Int(length / 2);

	for (i = 0; i < maxDepth + 1; i++) {
		if (threadIdx.x < threadMax) {
			l = threadIdx.x;
			r = l + threadMax;
			arrayIn[l] = arrayIn[r] + arrayIn[l];
		}
		threadMax = threadMax >> 1;
		__syncthreads(); // optimise -- use warp shuffle instead
	}

	if (threadIdx.x == 0) reducedArray[blockIdx.x] = arrayIn[0];
}


__device__ void reduceBlock(qreal *arrayIn, qreal *reducedArray, int length){
	unsigned int tid = threadIdx.x;

	//if (length >= 2048 && tid < 1024) arrayIn[tid] += arrayIn[tid + 1024];
	//__syncthreads();
	//if (length >= 1024 && tid < 512) arrayIn[tid] += arrayIn[tid + 512];
	//__syncthreads();
	if (length >= 512 && tid < 256) arrayIn[tid] += arrayIn[tid + 256];
	__syncthreads();
	if (length >= 256 && tid < 128) arrayIn[tid] += arrayIn[tid + 128];
	__syncthreads();
	if (length >= 128 && tid < 64) arrayIn[tid] += arrayIn[tid + 64];
	__syncthreads();
	if (length >= 64 && tid < 32) arrayIn[tid] += arrayIn[tid + 32];
	__syncthreads();

	if (length < 32)
	{
		reduceBlock1(arrayIn, reducedArray, length);
		return;
	}
	else
	{
		qreal localSum = arrayIn[tid];
		localSum += __shfl_xor(localSum, 16);
		localSum += __shfl_xor(localSum, 8);
		localSum += __shfl_xor(localSum, 4);
		localSum += __shfl_xor(localSum, 2);
		localSum += __shfl_xor(localSum, 1);

		if (tid == 0) reducedArray[blockIdx.x] = localSum;
	}

}


__global__ void copySharedReduceBlock(qreal*arrayIn, qreal *reducedArray, int length){
    extern __shared__ qreal tempReductionArray[];
    int blockOffset = blockIdx.x*length;
    tempReductionArray[threadIdx.x] = arrayIn[blockOffset + threadIdx.x];
   // tempReductionArray[threadIdx.x*2+1] = arrayIn[blockOffset + threadIdx.x*2+1];
    __syncthreads();
    reduceBlock(tempReductionArray, reducedArray, length);
}