#include "hip/hip_runtime.h"
#ifdef DEV
//优化中的代码
#else
//原始代码
#endif

#include "hip/hip_runtime.h"
#include ""

# include "QuEST.h"
# include "QuEST_precision.h"
# include "QuEST_internal.h"    // purely to resolve getQuESTDefaultSeedKey
# include "mt19937ar.h"

# include <stdlib.h>
# include <stdio.h>
# include <math.h>

# include "QuEST.h"

# define REDUCE_SHARED_SIZE 512
# define DEBUG 0


#define THREADS_PER_CUDA_BLOCK 256
#define TASKS_PER_KERNEL_FUC 32
//__constant__ qreal recRoot2 = 1.0 / sqrt(2.0);//dynamic initialization is computed at runtime
__constant__ qreal recRoot2 = 0.7071067811865475;



#ifdef __cplusplus
extern "C" {
#endif

//hadamardKernel单gpu的多个测试版本
#ifdef DEV
	__global__ void statevec_hadamardKernel_v0(Qureg qureg, const int targetQubit) {
		// ----- sizes
		long long int sizeBlock,                                           // size of blocks
			sizeHalfBlock;                                       // size of blocks halved
	   // ----- indices
		long long int thisBlock,                                           // current block
			indexUp, indexLo;                                     // current index and corresponding index in lower half block

	   // ----- temp variables
		qreal   stateRealUp, stateRealLo,                             // storage for previous state values
			stateImagUp, stateImagLo;                             // (used in updates)
	 // ----- temp variables
		long long int thisTask;                                   // task based approach for expose loop with small granularity
		const long long int numTasks = qureg.numAmpsPerChunk >> 1;

		sizeHalfBlock = 1LL << targetQubit;                               // size of blocks halved
		sizeBlock = 2LL * sizeHalfBlock;                           // size of blocks

		qreal *stateVecReal = qureg.deviceStateVec.real;
		qreal *stateVecImag = qureg.deviceStateVec.imag;

		qreal recRoot2 = 1.0 / sqrt(2.0);

		thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		thisBlock = thisTask / sizeHalfBlock;
		indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		indexLo = indexUp + sizeHalfBlock;

		// store current state vector values in temp variables
		stateRealUp = stateVecReal[indexUp];
		stateImagUp = stateVecImag[indexUp];

		stateRealLo = stateVecReal[indexLo];
		stateImagLo = stateVecImag[indexLo];

		stateVecReal[indexUp] = recRoot2 * (stateRealUp + stateRealLo);
		stateVecImag[indexUp] = recRoot2 * (stateImagUp + stateImagLo);

		stateVecReal[indexLo] = recRoot2 * (stateRealUp - stateRealLo);
		stateVecImag[indexLo] = recRoot2 * (stateImagUp - stateImagLo);
	}

	void statevec_hadamard_v0(Qureg qureg, const int targetQubit)
	{
		int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);
		statevec_hadamardKernel_v0 << <CUDABlocks, THREADS_PER_CUDA_BLOCK >> > (qureg, targetQubit);
	}

	// 接收预计算的参数
	__global__ void statevec_hadamardKernel_v1(
		Qureg qureg,
		const long long int numTasks, 
		const long long int sizeHalfBlock, 
		const long long int sizeBlock
	) {
		qreal *stateVecReal = qureg.deviceStateVec.real;
		qreal *stateVecImag = qureg.deviceStateVec.imag;

		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		long long int thisBlock = thisTask / sizeHalfBlock;
		long long int indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		long long int indexLo = indexUp + sizeHalfBlock;

		
		stateRealUp = stateVecReal[indexUp];
		stateImagUp = stateVecImag[indexUp];

		stateRealLo = stateVecReal[indexLo];
		stateImagLo = stateVecImag[indexLo];

		stateVecReal[indexUp] = recRoot2 * (stateRealUp + stateRealLo);
		stateVecImag[indexUp] = recRoot2 * (stateImagUp + stateImagLo);

		stateVecReal[indexLo] = recRoot2 * (stateRealUp - stateRealLo);
		stateVecImag[indexLo] = recRoot2 * (stateImagUp - stateImagLo);
	}

	void statevec_hadamard_v1(Qureg qureg, const int targetQubit)
	{
		const long long int numTasks = qureg.numAmpsPerChunk >> 1;
		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;

		int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);
		statevec_hadamardKernel_v1 << <CUDABlocks, THREADS_PER_CUDA_BLOCK >> > (
			qureg, numTasks,sizeHalfBlock,sizeBlock);
	}
	
	// 一个核函数处理 THREADS_PER_CUDA_BLOCK(2,4,8,...) 个任务
	__global__ void statevec_hadamardKernel_v2(
		Qureg qureg,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock
	) {
		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;

		qreal *stateVecReal = qureg.deviceStateVec.real;
		qreal *stateVecImag = qureg.deviceStateVec.imag;


		long long int thisTask = (blockIdx.x*blockDim.x + threadIdx.x)*TASKS_PER_KERNEL_FUC;
		/*
		https://devblogs.nvidia.com/how-access-global-memory-efficiently-cuda-c-kernels/
		For large strides, the effective bandwidth is poor regardless of architecture version.
		This should not be surprising:
		when concurrent threads simultaneously access memory addresses 
		that are very far apart in physical memory,
		then there is no chance for the hardware to combine the accesses.
		*/
		if (thisTask >= numTasks) return;

		long long int thisBlock = thisTask / sizeHalfBlock;
		long long int indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		long long int indexLo = indexUp + sizeHalfBlock;

		for (int i = 0; i < TASKS_PER_KERNEL_FUC; i++) {
			stateRealUp = stateVecReal[indexUp + i];
			stateImagUp = stateVecImag[indexUp + i];

			stateRealLo = stateVecReal[indexLo + i];
			stateImagLo = stateVecImag[indexLo + i];

			stateVecReal[indexUp + i] = recRoot2 * (stateRealUp + stateRealLo);
			stateVecImag[indexUp + i] = recRoot2 * (stateImagUp + stateImagLo);

			stateVecReal[indexLo + i] = recRoot2 * (stateRealUp - stateRealLo);
			stateVecImag[indexLo + i] = recRoot2 * (stateImagUp - stateImagLo);
		}
		
	}
	void statevec_hadamard_v2(Qureg qureg, const int targetQubit)
	{
		const long long int numTasks = qureg.numAmpsPerChunk >> 1;
		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;
		if (sizeHalfBlock < TASKS_PER_KERNEL_FUC) {
			int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);
			statevec_hadamardKernel_v0 << <CUDABlocks, THREADS_PER_CUDA_BLOCK >> > (qureg, targetQubit);
		}
		else {
			int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / (TASKS_PER_KERNEL_FUC*THREADS_PER_CUDA_BLOCK));
			statevec_hadamardKernel_v2 << <CUDABlocks, THREADS_PER_CUDA_BLOCK >> > (qureg, numTasks, sizeHalfBlock, sizeBlock);
		}
		
	}

	// 使用线程块共享的共享内存
	__global__ void statevec_hadamardKernel_v3(
		Qureg qureg,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock
	) {
		__shared__ qreal slice[4][THREADS_PER_CUDA_BLOCK];

		qreal *stateVecReal = qureg.deviceStateVec.real;
		qreal *stateVecImag = qureg.deviceStateVec.imag;


		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		long long int thisBlock = thisTask / sizeHalfBlock;
		long long int indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		long long int indexLo = indexUp + sizeHalfBlock;

		/*
		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;
		stateRealUp = stateVecReal[indexUp];
		stateImagUp = stateVecImag[indexUp];
		stateRealLo = stateVecReal[indexLo];
		stateImagLo = stateVecImag[indexLo];
		*/
		slice[0][threadIdx.x] = stateVecReal[indexUp];
		slice[1][threadIdx.x] = stateVecImag[indexUp];
		slice[2][threadIdx.x] = stateVecReal[indexLo];
		slice[3][threadIdx.x] = stateVecImag[indexLo];

		__syncthreads();

		stateVecReal[indexUp] = recRoot2 * (slice[0][threadIdx.x] + slice[2][threadIdx.x]);
		stateVecImag[indexUp] = recRoot2 * (slice[1][threadIdx.x] + slice[3][threadIdx.x]);

		stateVecReal[indexLo] = recRoot2 * (slice[0][threadIdx.x] - slice[2][threadIdx.x]);
		stateVecImag[indexLo] = recRoot2 * (slice[1][threadIdx.x] - slice[3][threadIdx.x]);
	}

	void statevec_hadamard_v3(Qureg qureg, const int targetQubit)
	{
		const long long int numTasks = qureg.numAmpsPerChunk >> 1;
		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;

		int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);
		if (sizeHalfBlock < THREADS_PER_CUDA_BLOCK) {
			statevec_hadamardKernel_v0 << <CUDABlocks, THREADS_PER_CUDA_BLOCK >> > (qureg, targetQubit);
		}
		else {
			statevec_hadamardKernel_v3 << <CUDABlocks, THREADS_PER_CUDA_BLOCK,
				4* THREADS_PER_CUDA_BLOCK*sizeof(qreal)>> > (
					qureg, numTasks, sizeHalfBlock, sizeBlock);
		}
	}

#endif

//多gpu支持
#ifdef DEV
	void setDevice(Qureg *qureg, int deviceID) {
		if (deviceID >= qureg->numChunks || deviceID<0) {
			exit(EXIT_FAILURE);
		}
		qureg->chunkId = deviceID;
		qureg->deviceStateVec = qureg->deviceStateVecList[deviceID];
		qureg->devicePairStateVec = qureg->devicePairStateVecList[deviceID];
		qureg->firstLevelReduction = qureg->firstLevelReductionList[deviceID];
		qureg->secondLevelReduction = qureg->secondLevelReductionList[deviceID];
		CUDA_CALL(hipSetDevice(deviceID));
		//printf("setting qureg on device %d, deviceStateVec.real %x\n", deviceID, qureg->deviceStateVec.real);
	}
	
	void sycAllDevices(Qureg qureg) {
		int originSeviceID = qureg.chunkId;
		for (int i = 0; i < qureg.numChunks; i++)
		{
			CUDA_CALL(hipSetDevice(i));
			CUDA_CALL(hipDeviceSynchronize());
		}
		setDevice(&qureg, originSeviceID);
	}

	void getDeviceArrHead(qreal *deviceArr) {
		qreal hostArr[HEAD_SIZE];
		hipMemcpy(hostArr, deviceArr, HEAD_SIZE * sizeof(qreal), hipMemcpyDeviceToHost);
		printf("device arr from %x is\n", deviceArr);
		for (int i = 0; i < HEAD_SIZE; ++i) {
			printf("%lf  ", hostArr[i]);
		}
		printf("\n");
	}

	void getHeadAmp(Qureg qureg) {
		getDeviceArrHead(qureg.deviceStateVecList[0].real);
		//getDeviceArrHead(qureg.deviceStateVecList[0].imag);
		getDeviceArrHead(qureg.deviceStateVecList[1].real);
		//getDeviceArrHead(qureg.deviceStateVecList[0].imag);
	}
#endif // DEV

//hadmand
#ifdef DEV
	__global__ void statevec_hadamardKernel_localGPU(
		qreal *stateVecReal,
		qreal *stateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock
	) {
		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		long long int thisBlock = thisTask / sizeHalfBlock;
		long long int indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		long long int indexLo = indexUp + sizeHalfBlock;


		stateRealUp = stateVecReal[indexUp];
		stateImagUp = stateVecImag[indexUp];

		stateRealLo = stateVecReal[indexLo];
		stateImagLo = stateVecImag[indexLo];

		stateVecReal[indexUp] = recRoot2 * (stateRealUp + stateRealLo);
		stateVecImag[indexUp] = recRoot2 * (stateImagUp + stateImagLo);

		stateVecReal[indexLo] = recRoot2 * (stateRealUp - stateRealLo);
		stateVecImag[indexLo] = recRoot2 * (stateImagUp - stateImagLo);
	}
	// 多GPU版本，支持访问外部（其他GPU）的数据
	__global__ void statevec_hadamardKernel_outerData(
		qreal *stateVecReal,
		qreal *stateVecImag,
		qreal *pairStateVecReal,
		qreal *pairStateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock,
		const int isLoChunk
	) {

		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		if (isLoChunk==0) {
			stateRealUp = stateVecReal[thisTask];
			stateImagUp = stateVecImag[thisTask];

			stateRealLo = pairStateVecReal[thisTask];
			stateImagLo = pairStateVecImag[thisTask];

			stateVecReal[thisTask] = recRoot2 * (stateRealUp + stateRealLo);
			stateVecImag[thisTask] = recRoot2 * (stateImagUp + stateImagLo);
		}
		else {
			stateRealLo = stateVecReal[thisTask];
			stateImagLo = stateVecImag[thisTask];

			stateRealUp = pairStateVecReal[thisTask];
			stateImagUp = pairStateVecImag[thisTask];

			stateVecReal[thisTask] = recRoot2 * (stateRealUp - stateRealLo);
			stateVecImag[thisTask] = recRoot2 * (stateImagUp - stateImagLo);
		}

	}
	void statevec_hadamard_multiGPU(Qureg qureg, const int targetQubit) {

		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;
		long long int CUDABlocks;
		int deviceID = qureg.chunkId;
		if (sizeBlock <= qureg.numAmpsPerChunk) {//data on local gpu is enough for blochsize
			const long long int numTasks = qureg.numAmpsPerChunk >> 1;
			CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);

			statevec_hadamardKernel_localGPU << <
				CUDABlocks, THREADS_PER_CUDA_BLOCK
				>> > (qureg.deviceStateVec.real, qureg.deviceStateVec.imag, numTasks, sizeHalfBlock, sizeBlock);

		}
		else
		{//data on local gpu is not enough for blochsize
			int halfChunkPerBlock = sizeHalfBlock / qureg.numAmpsPerChunk;
			int chunkPerBlock = 2 * halfChunkPerBlock;
			int isLoChunk = ((deviceID % chunkPerBlock) >= halfChunkPerBlock);
			int outerChunkID;
			qreal* dstReal;
			qreal* dstImag;
			qreal* srcReal;
			qreal* srcImag;
			if (!isLoChunk) {
				outerChunkID = deviceID + halfChunkPerBlock;
			}
			//由于使用了hipMemcpy，使用多个流
			if (!isLoChunk) {
				//使用多流将两个设备deviceID，outerChunkID的任务都完成，之后isLoChunk为True时不做任何工作
				long long int ampsPerStream = qureg.numAmpsPerChunk / USE_STREAM;
				CUDABlocks = ceil((qreal)(ampsPerStream / THREADS_PER_CUDA_BLOCK));
				for (int i = 0; i < USE_STREAM; i++) {
					setDevice(&qureg, deviceID);
					//prepare data for work on GPU deviceID
					dstReal = qureg.devicePairStateVec.real;
					dstImag = qureg.devicePairStateVec.imag;
					srcReal = qureg.deviceStateVecList[outerChunkID].real;
					srcImag = qureg.deviceStateVecList[outerChunkID].imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));

					//prepare data for work on GPU outerChunkID
					dstReal = qureg.devicePairStateVecList[outerChunkID].real;
					dstImag = qureg.devicePairStateVecList[outerChunkID].imag;
					srcReal = qureg.deviceStateVec.real;
					srcImag = qureg.deviceStateVec.imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));
					
					sycAllDevices(qureg);

					//allocate work on GPU deviceID
					statevec_hadamardKernel_outerData << <
						CUDABlocks, THREADS_PER_CUDA_BLOCK, 0,qureg.streamList[deviceID][i] >> > (
							qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 0);

					//allocate work on GPU outerChunkID
					setDevice(&qureg, outerChunkID);
					statevec_hadamardKernel_outerData << <
						CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[outerChunkID][i] >> > (
							qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 1);
				}
			}
		}
	}
#endif // DEV

//CNOT
#ifdef DEV
	static __device__ int extractBit(int locationOfBitFromRight, long long int theEncodedNumber)
	{
		return (theEncodedNumber & (1LL << locationOfBitFromRight)) >> locationOfBitFromRight;
	}
	__global__ void statevec_controlledNotKernel_localGPU(
		Qureg qureg, const int controlQubit,
		const long long int numTasks, 
		const long long int sizeHalfBlock, 
		const long long int sizeBlock)
	{
		// store current state vector values in temp variables
		qreal   stateRealUp, stateImagUp;
		long long int thisBlock, indexUp, indexLo, globalIndexUp;
		qreal *stateVecReal = qureg.deviceStateVec.real;
		qreal *stateVecImag = qureg.deviceStateVec.imag;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;
		thisBlock = thisTask / sizeHalfBlock;
		indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		indexLo = indexUp + sizeHalfBlock;

		globalIndexUp = qureg.chunkId*qureg.numAmpsPerChunk + indexUp;
		int controlBit = extractBit(controlQubit, globalIndexUp);
		if (controlBit) {
			stateRealUp = stateVecReal[indexUp];
			stateImagUp = stateVecImag[indexUp];

			stateVecReal[indexUp] = stateVecReal[indexLo];
			stateVecImag[indexUp] = stateVecImag[indexLo];

			stateVecReal[indexLo] = stateRealUp;
			stateVecImag[indexLo] = stateImagUp;
		}
	}
	__global__ void statevec_controlledNotKernel_outerData(
		Qureg qureg, const int controlQubit,
		qreal *stateVecReal,
		qreal *stateVecImag,
		qreal *pairStateVecReal,
		qreal *pairStateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock,
		const int isLoChunk
	) {
		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;
		long long int globalIndexUp = qureg.chunkId*qureg.numAmpsPerChunk + thisTask;
		int controlBit = extractBit(controlQubit, globalIndexUp);
		if (controlBit) {
			stateVecReal[thisTask] = pairStateVecReal[thisTask];
			stateVecImag[thisTask] = pairStateVecImag[thisTask];
		}
	}
	void statevec_controlledNot_multiGPU(Qureg qureg, const int controlQubit, const int targetQubit)
	{
		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;
		int deviceID = qureg.chunkId;
		if (sizeBlock <= qureg.numAmpsPerChunk) {
			const long long int numTasks = qureg.numAmpsPerChunk >> 1;
			int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);
			statevec_controlledNotKernel_localGPU << <CUDABlocks, THREADS_PER_CUDA_BLOCK >> > (
				qureg, controlQubit, numTasks, sizeHalfBlock, sizeBlock);
		}
		else {
			int halfChunkPerBlock = sizeHalfBlock / qureg.numAmpsPerChunk;
			int chunkPerBlock = 2 * halfChunkPerBlock;
			int isLoChunk = ((deviceID % chunkPerBlock) >= halfChunkPerBlock);
			int outerChunkID;
			qreal* dstReal;
			qreal* dstImag;
			qreal* srcReal;
			qreal* srcImag;
			if (isLoChunk) {
				outerChunkID = deviceID - halfChunkPerBlock;

				long long int ampsPerStream = qureg.numAmpsPerChunk / USE_STREAM;
				int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk / THREADS_PER_CUDA_BLOCK / USE_STREAM));
				
				for (int i = 0; i < USE_STREAM; i++) {
					setDevice(&qureg, deviceID);
					//prepare data for work on GPU deviceID
					dstReal = qureg.devicePairStateVec.real;
					dstImag = qureg.devicePairStateVec.imag;
					srcReal = qureg.deviceStateVecList[outerChunkID].real;
					srcImag = qureg.deviceStateVecList[outerChunkID].imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));

					//prepare data for work on GPU outerChunkID
					dstReal = qureg.devicePairStateVecList[outerChunkID].real;
					dstImag = qureg.devicePairStateVecList[outerChunkID].imag;
					srcReal = qureg.deviceStateVec.real;
					srcImag = qureg.deviceStateVec.imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));

					//both memcpy process should finish before kernel function is called
					sycAllDevices(qureg);

					//allocate work on GPU deviceID
					statevec_controlledNotKernel_outerData << <
						CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[deviceID][i] >> > (
							qureg, controlQubit,
							qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 0);

					//allocate work on GPU outerChunkID
					setDevice(&qureg, outerChunkID);
					statevec_controlledNotKernel_outerData << <
						CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[outerChunkID][i] >> > (
							qureg, controlQubit,
							qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 1);
				}
			}
		}
	}

#endif // DEV

//controlledCompactUnitary
#ifdef DEV
	//multiGpu implementation
	__global__ void statevec_compactUnitaryKernel_localGPU(
		Complex alpha, Complex beta,
		qreal *stateVecReal,
		qreal *stateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock)
	{

		qreal stateRealUp, stateRealLo,
			stateImagUp, stateImagLo,
			betaReal, betaImag,
			alphaImag, alphaReal;

		long long int thisTask = blockIdx.x * blockDim.x + threadIdx.x;
		if (thisTask >= numTasks)
			return;

		betaReal = beta.real;
		betaImag = beta.imag;
		alphaReal = alpha.real;
		alphaImag = alpha.imag;

		long long int thisBlock = thisTask / sizeHalfBlock;
		long long int indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		long long int indexLo = indexUp + sizeHalfBlock;

		stateRealUp = stateVecReal[indexUp];
		stateImagUp = stateVecImag[indexUp];

		stateRealLo = stateVecReal[indexLo];
		stateImagLo = stateVecImag[indexLo];

		stateVecReal[indexUp] = alphaReal * stateRealUp - alphaImag * stateImagUp - betaReal * stateRealLo - betaImag * stateImagLo;
		stateVecImag[indexUp] = alphaReal * stateImagUp + alphaImag * stateRealUp - betaReal * stateImagLo + betaImag * stateRealLo;

		// state[indexLo] = beta  * state[indexUp] + conj(alpha) * state[indexLo]
		stateVecReal[indexLo] = betaReal * stateRealUp - betaImag * stateImagUp + alphaReal * stateRealLo + alphaImag * stateImagLo;
		stateVecImag[indexLo] = betaReal * stateImagUp + betaImag * stateRealUp + alphaReal * stateImagLo - alphaImag * stateRealLo;
	}

	__global__ void statevec_compactUnitaryKernel_outerData(
		Complex alpha, Complex beta,
		qreal *stateVecReal,
		qreal *stateVecImag,
		qreal *pairStateVecReal,
		qreal *pairStateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock,
		const int isLoChunk)
	{
		qreal stateRealUp, stateRealLo,
			stateImagUp, stateImagLo,
			betaReal, betaImag,
			alphaImag, alphaReal;

		long long int thisTask = blockIdx.x * blockDim.x + threadIdx.x;
		if (thisTask >= numTasks)
			return;

		betaReal = beta.real;
		betaImag = beta.imag;
		alphaReal = alpha.real;
		alphaImag = alpha.imag;

		if (isLoChunk == 0)
		{
			stateRealUp = stateVecReal[thisTask];
			stateImagUp = stateVecImag[thisTask];

			stateRealLo = pairStateVecReal[thisTask];
			stateImagLo = pairStateVecImag[thisTask];

			stateVecReal[thisTask] = alphaReal * stateRealUp - alphaImag * stateImagUp - betaReal * stateRealLo - betaImag * stateImagLo;
			stateVecImag[thisTask] = alphaReal * stateImagUp + alphaImag * stateRealUp - betaReal * stateImagLo + betaImag * stateRealLo;
		}
		else
		{
			stateRealLo = stateVecReal[thisTask];
			stateImagLo = stateVecImag[thisTask];

			stateRealUp = pairStateVecReal[thisTask];
			stateImagUp = pairStateVecImag[thisTask];

			stateVecReal[thisTask] = betaReal * stateRealUp - betaImag * stateImagUp + alphaReal * stateRealLo + alphaImag * stateImagLo;
			stateVecImag[thisTask] = betaReal * stateImagUp + betaImag * stateRealUp + alphaReal * stateImagLo - alphaImag * stateRealLo;
		}
	}
	void statevec_compactUnitary_multiGPU(Qureg qureg, const int rotQubit, Complex alpha, Complex beta)
	{
		const long long int sizeHalfBlock = 1LL << rotQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;
		int deviceID = qureg.chunkId;
		if (sizeBlock <= qureg.numAmpsPerChunk)
		{ //data on local gpu is enough for blochsize
			const long long int numTasks = qureg.numAmpsPerChunk >> 1;
			int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);

			statevec_compactUnitaryKernel_localGPU << <
				CUDABlocks, THREADS_PER_CUDA_BLOCK >> > (alpha, beta, qureg.deviceStateVec.real,
					qureg.deviceStateVec.imag, numTasks, sizeHalfBlock, sizeBlock);
		}
		else
		{ //data on local gpu is not enough for blochsize
			int halfChunkPerBlock = sizeHalfBlock / qureg.numAmpsPerChunk;
			int chunkPerBlock = 2 * halfChunkPerBlock;
			int isLoChunk = ((deviceID % chunkPerBlock) >= halfChunkPerBlock); //numchunks 即为设备的数量
			int outerChunkID;
			qreal *dstReal;
			qreal *dstImag;
			qreal *srcReal;
			qreal *srcImag;

			if (!isLoChunk)
			{
				outerChunkID = deviceID + halfChunkPerBlock;

				//使用多流将两个设备deviceID，outerChunkID的任务都完成，之后isLoChunk为True时不做任何工作
				long long int ampsPerStream = qureg.numAmpsPerChunk / USE_STREAM;
				int CUDABlocks = ceil((qreal)(ampsPerStream / THREADS_PER_CUDA_BLOCK));
				for (int i = 0; i < USE_STREAM; i++)
				{
					setDevice(&qureg, deviceID);
					//prepare data for work on GPU deviceID
					dstReal = qureg.devicePairStateVec.real;
					dstImag = qureg.devicePairStateVec.imag;
					srcReal = qureg.deviceStateVecList[outerChunkID].real;
					srcImag = qureg.deviceStateVecList[outerChunkID].imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));

					//prepare data for work on GPU outerChunkID
					dstReal = qureg.devicePairStateVecList[outerChunkID].real;
					dstImag = qureg.devicePairStateVecList[outerChunkID].imag;
					srcReal = qureg.deviceStateVec.real;
					srcImag = qureg.deviceStateVec.imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));

					sycAllDevices(qureg);

					//allocate work on GPU deviceID
					statevec_compactUnitaryKernel_outerData << <
						CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[deviceID][i] >> > (alpha, beta,
							qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 0);

					//allocate work on GPU outerChunkID
					setDevice(&qureg, outerChunkID);
					statevec_compactUnitaryKernel_outerData << <
						CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[outerChunkID][i] >> > (alpha, beta,
							qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 1);
				}
			}
		}
	}
#endif // DEV

//controlledCompactUnitary
#ifdef DEV

	__global__ void statevec_controlledCompactUnitaryKernel_localGPU(
		Qureg qureg, const int controlQubit,
		Complex alpha, Complex beta,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock)
	{
		qreal stateRealUp, stateImagUp, stateRealLo, stateImagLo,
			alphaImag, alphaReal, betaReal, betaImag;
		long long int thisBlock, indexUp, indexLo, globalIndexUp;
		qreal *stateVecReal = qureg.deviceStateVec.real;
		qreal *stateVecImag = qureg.deviceStateVec.imag;

		betaReal = beta.real;
		betaImag = beta.imag;
		alphaReal = alpha.real;
		alphaImag = alpha.imag;

		long long int thisTask = blockIdx.x * blockDim.x + threadIdx.x;
		if (thisTask >= numTasks)
			return;
		thisBlock = thisTask / sizeHalfBlock;
		indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		indexLo = indexUp + sizeHalfBlock;

		globalIndexUp = qureg.chunkId * qureg.numAmpsPerChunk + indexUp;
		int controlBit = extractBit(controlQubit, globalIndexUp);

		if (controlBit)
		{
			// store current state vector values in temp variables
			stateRealUp = stateVecReal[indexUp];
			stateImagUp = stateVecImag[indexUp];

			stateRealLo = stateVecReal[indexLo];
			stateImagLo = stateVecImag[indexLo];

			// state[indexUp] = alpha * state[indexUp] - conj(beta)  * state[indexLo]
			stateVecReal[indexUp] = alphaReal * stateRealUp - alphaImag * stateImagUp - betaReal * stateRealLo - betaImag * stateImagLo;
			stateVecImag[indexUp] = alphaReal * stateImagUp + alphaImag * stateRealUp - betaReal * stateImagLo + betaImag * stateRealLo;

			// state[indexLo] = beta  * state[indexUp] + conj(alpha) * state[indexLo]
			stateVecReal[indexLo] = betaReal * stateRealUp - betaImag * stateImagUp + alphaReal * stateRealLo + alphaImag * stateImagLo;
			stateVecImag[indexLo] = betaReal * stateImagUp + betaImag * stateRealUp + alphaReal * stateImagLo - alphaImag * stateRealLo;
		}
	}
	__global__ void statevec_controlledCompactUnitaryKernel_outerData(
		Qureg qureg, const int controlQubit,
		Complex alpha, Complex beta,
		qreal *stateVecReal,
		qreal *stateVecImag,
		qreal *pairStateVecReal,
		qreal *pairStateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock,
		const int isLoChunk)
	{

		qreal stateRealUp, stateRealLo,
			stateImagUp, stateImagLo,
			betaReal, betaImag,
			alphaReal, alphaImag;

		long long int thisTask = blockIdx.x * blockDim.x + threadIdx.x;
		if (thisTask >= numTasks)
			return;

		betaReal = beta.real;
		betaImag = beta.imag;
		alphaReal = alpha.real;
		alphaImag = alpha.imag;


		long long int globalIndexUp = qureg.chunkId * qureg.numAmpsPerChunk + thisTask;
		int controlBit = extractBit(controlQubit, globalIndexUp);
		if (controlBit)
		{
			if (isLoChunk == 0)
			{

				stateRealUp = stateVecReal[thisTask];
				stateImagUp = stateVecImag[thisTask];

				stateRealLo = pairStateVecReal[thisTask];
				stateImagLo = pairStateVecImag[thisTask];

				stateVecReal[thisTask] = alphaReal * stateRealUp - alphaImag * stateImagUp - betaReal * stateRealLo - betaImag * stateImagLo;

				stateVecImag[thisTask] = alphaReal * stateImagUp + alphaImag * stateRealUp - betaReal * stateImagLo + betaImag * stateRealLo;
			}
			else
			{
				stateRealLo = stateVecReal[thisTask];
				stateImagLo = stateVecImag[thisTask];

				stateRealUp = pairStateVecReal[thisTask];
				stateImagUp = pairStateVecImag[thisTask];

				stateVecReal[thisTask] = betaReal * stateRealUp - betaImag * stateImagUp + alphaReal * stateRealLo + alphaImag * stateImagLo;

				stateVecImag[thisTask] = betaReal * stateImagUp + betaImag * stateRealUp + alphaReal * stateImagLo - alphaImag * stateRealLo;
			}
		}
	}


	void statevec_controlledCompactUnitary_multiGPU(
		Qureg qureg, const int controlQubit, const int targetQubit,
		Complex alpha, Complex beta)
	{
		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;
		int deviceID = qureg.chunkId;
		if (sizeBlock <= qureg.numAmpsPerChunk)
		{ //data on local gpu is enough for blochsize
			const long long int numTasks = qureg.numAmpsPerChunk >> 1;
			int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);

			statevec_controlledCompactUnitaryKernel_localGPU << <
				CUDABlocks, THREADS_PER_CUDA_BLOCK >> > (qureg, controlQubit, alpha, beta,
					numTasks, sizeHalfBlock, sizeBlock);
		}
		else
		{ //data on local gpu is not enough for blochsize
			int halfChunkPerBlock = sizeHalfBlock / qureg.numAmpsPerChunk;
			int chunkPerBlock = 2 * halfChunkPerBlock;
			int isLoChunk = ((deviceID % chunkPerBlock) >= halfChunkPerBlock); //numchunks 即为设备的数量
			int outerChunkID;
			qreal *dstReal;
			qreal *dstImag;
			qreal *srcReal;
			qreal *srcImag;
			if (!isLoChunk)
			{
				outerChunkID = deviceID + halfChunkPerBlock;
			}
			if (!isLoChunk)
			{
				long long int ampsPerStream = qureg.numAmpsPerChunk / USE_STREAM;
				int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk / THREADS_PER_CUDA_BLOCK / USE_STREAM));

				for (int i = 0; i < USE_STREAM; i++)
				{
					setDevice(&qureg, deviceID);
					//prepare data for work on GPU deviceID
					dstReal = qureg.devicePairStateVec.real;
					dstImag = qureg.devicePairStateVec.imag;
					srcReal = qureg.deviceStateVecList[outerChunkID].real;
					srcImag = qureg.deviceStateVecList[outerChunkID].imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));

					//prepare data for work on GPU outerChunkID
					dstReal = qureg.devicePairStateVecList[outerChunkID].real;
					dstImag = qureg.devicePairStateVecList[outerChunkID].imag;
					srcReal = qureg.deviceStateVec.real;
					srcImag = qureg.deviceStateVec.imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));

					sycAllDevices(qureg);

					//allocate work on GPU deviceID
					statevec_controlledCompactUnitaryKernel_outerData << <CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[deviceID][i] >> >
						(qureg, controlQubit, alpha, beta,
							qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 0);

					//allocate work on GPU outerChunkID
					setDevice(&qureg, outerChunkID);
					statevec_controlledCompactUnitaryKernel_outerData << <CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[outerChunkID][i] >> >
						(qureg, controlQubit, alpha, beta,
							qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 1);
				}
			}
		}
	}
#endif // DEV

//phaseShiftByTerm
#ifdef DEV

	__global__ void statevec_phaseShiftByTermKernel_localGPU(
		qreal *stateVecReal,
		qreal *stateVecImag,
		qreal cosAngle, qreal sinAngle,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock
	) {
		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		long long int thisBlock = thisTask / sizeHalfBlock;
		long long int indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		long long int indexLo = indexUp + sizeHalfBlock;

		stateRealLo = stateVecReal[indexLo];
		stateImagLo = stateVecImag[indexLo];

		stateVecReal[indexLo] = cosAngle * stateRealLo - sinAngle * stateImagLo;
		stateVecImag[indexLo] = sinAngle * stateRealLo + cosAngle * stateImagLo;
	}
	__global__ void statevec_phaseShiftByTerm_outerData(
		qreal *stateVecReal,
		qreal *stateVecImag,
		qreal cosAngle, qreal sinAngle,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock)
	{

		qreal stateRealUp, stateRealLo,
			stateImagUp, stateImagLo,
			betaReal, betaImag,
			alphaReal, alphaImag;

		long long int thisTask = blockIdx.x * blockDim.x + threadIdx.x;
		if (thisTask >= numTasks)
			return;

		stateRealLo = stateVecReal[thisTask];
		stateImagLo = stateVecImag[thisTask];

		stateVecReal[thisTask] = cosAngle * stateRealLo - sinAngle * stateImagLo;
		stateVecImag[thisTask] = sinAngle * stateRealLo + cosAngle * stateImagLo;
		
	}
	
	void statevec_phaseShiftByTerm_multiGPU(Qureg qureg, const int targetQubit, Complex term) {

		qreal cosAngle = term.real;
		qreal sinAngle = term.imag;
		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;
		int deviceID = qureg.chunkId;
		if (sizeBlock <= qureg.numAmpsPerChunk) {//data on local gpu is enough for blochsize
			const long long int numTasks = qureg.numAmpsPerChunk >> 1;
			int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);

			statevec_phaseShiftByTermKernel_localGPU << <CUDABlocks, THREADS_PER_CUDA_BLOCK>> > (
				qureg.deviceStateVec.real, qureg.deviceStateVec.imag,
				cosAngle, sinAngle,
				numTasks, sizeHalfBlock, sizeBlock);

		}
		else
		{
			int halfChunkPerBlock = sizeHalfBlock / qureg.numAmpsPerChunk;
			int chunkPerBlock = 2 * halfChunkPerBlock;
			int isLoChunk = ((deviceID % chunkPerBlock) >= halfChunkPerBlock);
			if (isLoChunk) {
				int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk / THREADS_PER_CUDA_BLOCK));
				
				//only stateVec[indexLo] is referenced and modified
				statevec_phaseShiftByTerm_outerData << <CUDABlocks, THREADS_PER_CUDA_BLOCK>> > (
					qureg.deviceStateVec.real, qureg.deviceStateVec.imag,
					cosAngle, sinAngle,
					qureg.numAmpsPerChunk, sizeHalfBlock, sizeBlock);
				
			}
		}
	}
#endif // DEV

//pauliX
#ifdef DEV
	__global__ void statevec_pauliXKernel_localGPU(
		qreal *stateVecReal,
		qreal *stateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock
	) {

		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		long long int thisBlock = thisTask / sizeHalfBlock;
		long long int indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		long long int indexLo = indexUp + sizeHalfBlock;

		// store current state vector values in temp variables
		stateRealUp = stateVecReal[indexUp];
		stateImagUp = stateVecImag[indexUp];

		stateVecReal[indexUp] = stateVecReal[indexLo];
		stateVecImag[indexUp] = stateVecImag[indexLo];

		stateVecReal[indexLo] = stateRealUp;
		stateVecImag[indexLo] = stateImagUp;
	}

	__global__ void statevec_pauliXKernel_outerData(
		qreal *stateVecReal,
		qreal *stateVecImag,
		qreal *pairStateVecReal,
		qreal *pairStateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock,
		const int isLoChunk
	) {

		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		if (isLoChunk == 0) {//indexUp
			stateRealLo = pairStateVecReal[thisTask];
			stateImagLo = pairStateVecImag[thisTask];

			stateVecReal[thisTask] = stateRealLo;
			stateVecImag[thisTask] = stateImagLo;
		}
		else {//indexLo

			stateRealUp = pairStateVecReal[thisTask];
			stateImagUp = pairStateVecImag[thisTask];

			stateVecReal[thisTask] = stateRealUp;
			stateVecImag[thisTask] = stateImagUp;
		}
	}

	void statevec_pauliX_multiGPU(Qureg qureg, const int targetQubit)
	{
		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;
		int deviceID = qureg.chunkId;
		if (sizeBlock <= qureg.numAmpsPerChunk)
		{ //data on local gpu is enough for blochsize
			const long long int numTasks = qureg.numAmpsPerChunk >> 1;
			int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);
			statevec_pauliXKernel_localGPU << <CUDABlocks, THREADS_PER_CUDA_BLOCK >> >
				(qureg.deviceStateVec.real, qureg.deviceStateVec.imag, numTasks, sizeHalfBlock, sizeBlock);
		}
		else
		{ //data on local gpu is not enough for blochsize
			int halfChunkPerBlock = sizeHalfBlock / qureg.numAmpsPerChunk;
			int chunkPerBlock = 2 * halfChunkPerBlock;
			int isLoChunk = ((deviceID % chunkPerBlock) >= halfChunkPerBlock); //numchunks 即为设备的数量
			int outerChunkID;
			qreal *dstReal;
			qreal *dstImag;
			qreal *srcReal;
			qreal *srcImag;
			if (!isLoChunk)
			{
				outerChunkID = deviceID + halfChunkPerBlock;
				long long int ampsPerStream = qureg.numAmpsPerChunk / USE_STREAM;
				int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk / THREADS_PER_CUDA_BLOCK / USE_STREAM));

				for (int i = 0; i < USE_STREAM; i++)
				{
					setDevice(&qureg, deviceID);
					//prepare data for work on GPU deviceID
					dstReal = qureg.devicePairStateVec.real;
					dstImag = qureg.devicePairStateVec.imag;
					srcReal = qureg.deviceStateVecList[outerChunkID].real;
					srcImag = qureg.deviceStateVecList[outerChunkID].imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));

					//prepare data for work on GPU outerChunkID
					dstReal = qureg.devicePairStateVecList[outerChunkID].real;
					dstImag = qureg.devicePairStateVecList[outerChunkID].imag;
					srcReal = qureg.deviceStateVec.real;
					srcImag = qureg.deviceStateVec.imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));

					sycAllDevices(qureg);

					//allocate work on GPU deviceID
					statevec_pauliXKernel_outerData << <CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[deviceID][i] >> >
						(qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 0);

					//allocate work on GPU outerChunkID
					setDevice(&qureg, outerChunkID);
					statevec_pauliXKernel_outerData << <CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[outerChunkID][i] >> >
						(qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 1);
				}
			}
		}

	}

#endif // DEV

//pauliY
#ifdef DEV
	__global__ void statevec_pauliYKernel_localGPU(
		const int conjFac,
		qreal *stateVecReal,
		qreal *stateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock
	) {

		qreal  stateRealUp, stateImagUp;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		long long int thisBlock = thisTask / sizeHalfBlock;
		long long int indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		long long int indexLo = indexUp + sizeHalfBlock;

		stateRealUp = stateVecReal[indexUp];
		stateImagUp = stateVecImag[indexUp];

		// update under +-{{0, -i}, {i, 0}}
		stateVecReal[indexUp] = conjFac * stateVecImag[indexLo];
		stateVecImag[indexUp] = conjFac * -stateVecReal[indexLo];
		stateVecReal[indexLo] = conjFac * -stateImagUp;
		stateVecImag[indexLo] = conjFac * stateRealUp;
	}
	__global__ void statevec_pauliYKernel_outerData(
		const int conjFac,
		qreal *stateVecReal,
		qreal *stateVecImag,
		qreal *pairStateVecReal,
		qreal *pairStateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock,
		const int isLoChunk
	) {

		qreal  stateRealUp, stateRealLo, stateImagUp, stateImagLo;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		if (isLoChunk == 0) {//indexUp
			stateRealLo = pairStateVecReal[thisTask];
			stateImagLo = pairStateVecImag[thisTask];

			stateVecReal[thisTask] = conjFac * stateImagLo;
			stateVecImag[thisTask] = conjFac * -stateRealLo;
		}
		else {//indexLo

			stateRealUp = pairStateVecReal[thisTask];
			stateImagUp = pairStateVecImag[thisTask];

			stateVecReal[thisTask] = conjFac * -stateImagUp;
			stateVecImag[thisTask] = conjFac * stateRealUp;
		}
	}


	void statevec_pauliY_multiGPU(Qureg qureg, const int targetQubit)
	{
		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;
		int deviceID = qureg.chunkId;
		if (sizeBlock <= qureg.numAmpsPerChunk)
		{ //data on local gpu is enough for blochsize
			const long long int numTasks = qureg.numAmpsPerChunk >> 1;
			int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);
			statevec_pauliYKernel_localGPU << <CUDABlocks, THREADS_PER_CUDA_BLOCK >> >
				(1, qureg.deviceStateVec.real, qureg.deviceStateVec.imag, numTasks, sizeHalfBlock, sizeBlock);
		}
		else
		{ //data on local gpu is not enough for blochsize
			int halfChunkPerBlock = sizeHalfBlock / qureg.numAmpsPerChunk;
			int chunkPerBlock = 2 * halfChunkPerBlock;
			int isLoChunk = ((deviceID % chunkPerBlock) >= halfChunkPerBlock); //numchunks 即为设备的数量
			int outerChunkID;
			qreal *dstReal;
			qreal *dstImag;
			qreal *srcReal;
			qreal *srcImag;
			if (!isLoChunk)
			{
				outerChunkID = deviceID + halfChunkPerBlock;
				long long int ampsPerStream = qureg.numAmpsPerChunk / USE_STREAM;
				int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk / THREADS_PER_CUDA_BLOCK / USE_STREAM));

				for (int i = 0; i < USE_STREAM; i++)
				{
					setDevice(&qureg, deviceID);
					//prepare data for work on GPU deviceID
					dstReal = qureg.devicePairStateVec.real;
					dstImag = qureg.devicePairStateVec.imag;
					srcReal = qureg.deviceStateVecList[outerChunkID].real;
					srcImag = qureg.deviceStateVecList[outerChunkID].imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));

					//prepare data for work on GPU outerChunkID
					dstReal = qureg.devicePairStateVecList[outerChunkID].real;
					dstImag = qureg.devicePairStateVecList[outerChunkID].imag;
					srcReal = qureg.deviceStateVec.real;
					srcImag = qureg.deviceStateVec.imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));

					sycAllDevices(qureg);

					//allocate work on GPU deviceID
					statevec_pauliYKernel_outerData << <CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[deviceID][i] >> >
						(1, qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 0);

					//allocate work on GPU outerChunkID
					setDevice(&qureg, outerChunkID);
					statevec_pauliYKernel_outerData << <CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[outerChunkID][i] >> >
						(1, qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 1);
				}
			}
		}

	}


#endif // DEV

//controlledPauliY
#ifdef DEV
	__global__ void statevec_controlledPauliYKernel_outerData(
		Qureg qureg,
		const int controlQubit,
		const int conjFac,
		qreal *stateVecReal,
		qreal *stateVecImag,
		qreal *pairStateVecReal,
		qreal *pairStateVecImag,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock,
		const int isLoChunk)
	{
		qreal stateRealUp, stateRealLo,
			stateImagUp, stateImagLo;

		long long int thisTask = blockIdx.x * blockDim.x + threadIdx.x;
		if (thisTask >= numTasks)
			return;

		long long int globalIndexUp = qureg.chunkId * qureg.numAmpsPerChunk + thisTask;
		int controlBit = extractBit(controlQubit, globalIndexUp);
		if (controlBit)
		{
			if (isLoChunk == 0)
			{
				stateRealLo = pairStateVecReal[thisTask];
				stateImagLo = pairStateVecImag[thisTask];

				stateVecReal[thisTask] = conjFac * stateImagLo;
				stateVecImag[thisTask] = conjFac * -stateRealLo;
			}
			else
			{

				stateRealUp = pairStateVecReal[thisTask];
				stateImagUp = pairStateVecImag[thisTask];

				stateVecReal[thisTask] = conjFac * -stateImagUp;
				stateVecImag[thisTask] = conjFac * stateRealUp;
			}
		}

	}

	__global__ void statevec_controlledPauliYKernel_localGPU(
		Qureg qureg,
		const int controlQubit,
		const int conjFac,
		const long long int numTasks,
		const long long int sizeHalfBlock,
		const long long int sizeBlock
	)
	{
		//qreal stateRealUp, stateImagUp, stateRealLo,stateImagLo;
		long long int thisBlock, indexUp, indexLo, globalIndexUp;
		int controlBit;

		qreal   stateRealUp, stateImagUp;

		qreal *stateVecReal = qureg.deviceStateVec.real;
		qreal *stateVecImag = qureg.deviceStateVec.imag;

		long long int thisTask = blockIdx.x*blockDim.x + threadIdx.x;
		if (thisTask >= numTasks) return;

		thisBlock = thisTask / sizeHalfBlock;
		indexUp = thisBlock * sizeBlock + thisTask % sizeHalfBlock;
		indexLo = indexUp + sizeHalfBlock;

		globalIndexUp = qureg.chunkId * qureg.numAmpsPerChunk + indexUp;
		controlBit = extractBit(controlQubit, globalIndexUp);
		if (controlBit) {

			stateRealUp = stateVecReal[indexUp];
			stateImagUp = stateVecImag[indexUp];

			// update under +-{{0, -i}, {i, 0}}
			stateVecReal[indexUp] = conjFac * stateVecImag[indexLo];
			stateVecImag[indexUp] = conjFac * -stateVecReal[indexLo];
			stateVecReal[indexLo] = conjFac * -stateImagUp;
			stateVecImag[indexLo] = conjFac * stateRealUp;
		}
	}

	void statevec_controlledPauliY_multiGPU(Qureg qureg, const int controlQubit, const int targetQubit)
	{
		const long long int sizeHalfBlock = 1LL << targetQubit;
		const long long int sizeBlock = 2LL * sizeHalfBlock;
		int deviceID = qureg.chunkId;
		if (sizeBlock <= qureg.numAmpsPerChunk)
		{ //data on local gpu is enough for blochsize
			const long long int numTasks = qureg.numAmpsPerChunk >> 1;
			int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk >> 1) / THREADS_PER_CUDA_BLOCK);
			statevec_controlledPauliYKernel_localGPU << <CUDABlocks, THREADS_PER_CUDA_BLOCK >> >
				(qureg, controlQubit, 1, numTasks, sizeHalfBlock, sizeBlock);
		}
		else
		{ //data on local gpu is not enough for blochsize
			int halfChunkPerBlock = sizeHalfBlock / qureg.numAmpsPerChunk;
			int chunkPerBlock = 2 * halfChunkPerBlock;
			int isLoChunk = ((deviceID % chunkPerBlock) >= halfChunkPerBlock); //numchunks 即为设备的数量
			int outerChunkID;
			qreal *dstReal;
			qreal *dstImag;
			qreal *srcReal;
			qreal *srcImag;
			if (!isLoChunk)
			{
				outerChunkID = deviceID + halfChunkPerBlock;
				long long int ampsPerStream = qureg.numAmpsPerChunk / USE_STREAM;
				int CUDABlocks = ceil((qreal)(qureg.numAmpsPerChunk / THREADS_PER_CUDA_BLOCK / USE_STREAM));

				for (int i = 0; i < USE_STREAM; i++)
				{
					setDevice(&qureg, deviceID);
					//prepare data for work on GPU deviceID
					dstReal = qureg.devicePairStateVec.real;
					dstImag = qureg.devicePairStateVec.imag;
					srcReal = qureg.deviceStateVecList[outerChunkID].real;
					srcImag = qureg.deviceStateVecList[outerChunkID].imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[deviceID][i]));

					//prepare data for work on GPU outerChunkID
					dstReal = qureg.devicePairStateVecList[outerChunkID].real;
					dstImag = qureg.devicePairStateVecList[outerChunkID].imag;
					srcReal = qureg.deviceStateVec.real;
					srcImag = qureg.deviceStateVec.imag;
					CUDA_CALL(hipMemcpyAsync(dstReal + i * ampsPerStream, srcReal + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));
					CUDA_CALL(hipMemcpyAsync(dstImag + i * ampsPerStream, srcImag + i * ampsPerStream,
						ampsPerStream * sizeof(qreal), hipMemcpyDefault, qureg.streamList[outerChunkID][i]));

					sycAllDevices(qureg);

					//allocate work on GPU deviceID
					statevec_controlledPauliYKernel_outerData << <CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[deviceID][i] >> >
						(qureg, controlQubit, 1, qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 0);

					//allocate work on GPU outerChunkID
					setDevice(&qureg, outerChunkID);
					statevec_controlledPauliYKernel_outerData << <CUDABlocks, THREADS_PER_CUDA_BLOCK, 0, qureg.streamList[outerChunkID][i] >> >
						(qureg, controlQubit, 1, qureg.deviceStateVec.real + i * ampsPerStream, qureg.deviceStateVec.imag + i * ampsPerStream,
							qureg.devicePairStateVec.real + i * ampsPerStream, qureg.devicePairStateVec.imag + i * ampsPerStream,
							ampsPerStream, sizeHalfBlock, sizeBlock, 1);
				}
			}
		}
	}

#endif // DEV

#ifdef __cplusplus
}
#endif
